#include "hip/hip_runtime.h"
// To Compile: nvcc fkmigCUDA.cu -o fkmigCUDA.out -I/usr/local/cuda/include -L/usr/local/cuda/lib -lcufft
// To Run: ./fkmigCUDA.out SIG.txt nt nx fs pitch TXangle c t0 migSIG.txt

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <typeinfo>
#include <iostream>
#include <mex.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <math.h>

// Define macros for excessively long names
#define pi 3.141592653589793238462643383279502884197169399375105820974f
#define CCE checkCudaErrors
#define HtoD hipMemcpyHostToDevice
#define DtoH hipMemcpyDeviceToHost

// CUDA Texture Objects for Real and Imaginary Parts of Spatiotemporal Frequency Domain of Signals
static hipTextureObject_t texObjReal;
static hipTextureDesc texDescReal;
static hipResourceDesc resDescReal;
static hipTextureObject_t texObjImag;
static hipTextureDesc texDescImag;
static hipResourceDesc resDescImag;

// Runs batched FFT and IFFT on device data
void batchedFFT(hipfftComplex* dData, int N, int BATCH) {
	hipfftHandle plan;
	if (hipfftPlan1d(&plan, N, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}
	if (hipfftExecC2C(plan, dData, dData, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
	}
	if (hipDeviceSynchronize() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
	}
}
void batchedIFFT(hipfftComplex* dData, int N, int BATCH) {
	hipfftHandle plan;
	if (hipfftPlan1d(&plan, N, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}
	if (hipfftExecC2C(plan, dData, dData, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
	}
	if (hipDeviceSynchronize() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
	}
}

// Outputs Matrix Transpose
__global__ void transpose(hipfftComplex *odata, hipfftComplex *idata, int numRows, int numCols)
{
	int c_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int r_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (r_idx < numRows && c_idx < numCols) {
		float origx = idata[c_idx + numCols * r_idx].x;
		float origy = idata[c_idx + numCols * r_idx].y;
		__syncthreads();
		odata[r_idx + numRows * c_idx].x = origx;
		odata[r_idx + numRows * c_idx].y = origy;
	}
}

// Trim the RF Signals
__global__ void rfTrim(hipfftComplex *SIG, int nf, int nx, int nxFFT, float *dt, float *f0, float t0)
{
	int x_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int f_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (x_idx < nx && f_idx < nf) {
		float realSIG = SIG[x_idx + f_idx * nxFFT].x;
		float imagSIG = SIG[x_idx + f_idx * nxFFT].y;
		float phase = -2 * pi*(dt[x_idx] + t0)*f0[f_idx];
		SIG[x_idx + f_idx * nxFFT].x = realSIG * cosf(phase) - imagSIG * sinf(phase);
		SIG[x_idx + f_idx * nxFFT].y = realSIG * sinf(phase) + imagSIG * cosf(phase);
	}
}

// Remove Evanescent Parts in Spatio-temporal Frequency Domain of the Signals
__global__ void removeEvanescent(hipfftComplex *SIG, float *f0, int nf, float *kx, int nxFFT, float c)
{
	int kx_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int f0_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (kx_idx < nxFFT && f0_idx < nf) {
		if (abs(f0[f0_idx]) * abs(kx[kx_idx]) < c) {
			SIG[kx_idx + f0_idx * nxFFT].x = 0;
			SIG[kx_idx + f0_idx * nxFFT].y = 0;
		}
	}
}

// Get Real and Imaginary Parts
__global__ void getRealAndImag(hipfftComplex *cmpdata, float *realdata, float *imagdata, int numRows, int numCols)
{
	int c_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int r_idx = blockIdx.y*blockDim.y + threadIdx.y;
	if (r_idx < numRows && c_idx < numCols) {
		realdata[c_idx + numCols * r_idx] = cmpdata[c_idx + numCols * r_idx].x;
		imagdata[c_idx + numCols * r_idx] = cmpdata[c_idx + numCols * r_idx].y;
	}
}

// Run Stolt Mapping Kernel
__global__ void stoltmap(hipfftComplex *SIG, float *f0, float *kx, int ntFFT, int nxFFT, float c, float v, float beta, float fs, hipTextureObject_t texObjReal, hipTextureObject_t texObjImag)
{
	int f0_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int kx_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (kx_idx == 0 && f0_idx == 0) {
		SIG[0].x = 0;
		SIG[0].y = 0;
	}
	else if (kx_idx < nxFFT && f0_idx < ntFFT / 2 + 1) {
		// Note: we choose kz = 2*f/c (i.e. z = c*t/2)
		float Kx = kx[kx_idx];
		float f = f0[f0_idx];
		float fkz = v*sqrt(Kx*Kx + 4 * ((f*f) / (c*c)) / (beta*beta));
		__syncthreads();
		// Linear interpolation in the frequency domain: f -> fkz
		float fkz_idx = (fkz / (fs / ntFFT)) + 0.5f;
		float SIGreal = tex1DLayered<float>(texObjReal, fkz_idx, kx_idx);
		float SIGimag = tex1DLayered<float>(texObjImag, fkz_idx, kx_idx);
		__syncthreads();
		// Multiply By Obliquity factor: f / fkz
		SIG[kx_idx + f0_idx * nxFFT].x = SIGreal * f / fkz;
		SIG[kx_idx + f0_idx * nxFFT].y = SIGimag * f / fkz;
		__syncthreads();
	}
}

// Concatenate Negative Axial Frequencies to Fourier Domain of Migrated Solution
__global__ void concatNegAxialFreq(hipfftComplex *SIG, hipfftComplex *SIGfromTexture, int ntFFT, int nxFFT)
{
	int kx_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int f0_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (kx_idx < nxFFT && f0_idx < ntFFT) {
		if (f0_idx < ntFFT / 2 + 1) {
			// Original Part
			SIG[kx_idx + f0_idx * nxFFT].x = SIGfromTexture[kx_idx + f0_idx * nxFFT].x;
			SIG[kx_idx + f0_idx * nxFFT].y = SIGfromTexture[kx_idx + f0_idx * nxFFT].y;
		}
		else {
			// Concatenated Part
			SIG[kx_idx + f0_idx * nxFFT].x = SIGfromTexture[((nxFFT - kx_idx) % nxFFT) + (ntFFT - f0_idx) * nxFFT].x;
			SIG[kx_idx + f0_idx * nxFFT].y = -SIGfromTexture[((nxFFT - kx_idx) % nxFFT) + (ntFFT - f0_idx) * nxFFT].y;
		}
	}
}

// Steering Angle Compensation for RF Signals
__global__ void steerComp(hipfftComplex *SIG, int nxFFT, int ntFFT, float *kx, float fs, float c, float gamma)
{
	int kx_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int t_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (kx_idx < nxFFT && t_idx < ntFFT) {
		float realSIG = SIG[kx_idx + t_idx * nxFFT].x;
		float imagSIG = SIG[kx_idx + t_idx * nxFFT].y;
		float dx = -gamma*t_idx / fs * c / 2;
		float phase = -2 * pi * kx[kx_idx] * dx;
		SIG[kx_idx + t_idx * nxFFT].x = realSIG * cosf(phase) - imagSIG * sinf(phase);
		SIG[kx_idx + t_idx * nxFFT].y = realSIG * sinf(phase) + imagSIG * cosf(phase);
	}
}


// Gateway function
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

	// Argument check
	if (nrhs != 6)	{ mexErrMsgTxt("Wrong number of inputs.\n"); }
  if (nlhs != 1)	{ mexErrMsgTxt("Wrong number of outputs.\n"); }

  // Gather values from inputs
  double *SIGinput = (double *)mxGetData(prhs[0]);
  const mwSize *dimsSIG = mxGetDimensions(prhs[0]);
  mwSize nx = dimsSIG[0]; // Number of Array Elements [ROWS]
  mwSize nt = dimsSIG[1]; // Number of Time Points [COLUMNS]
  float fs = mxGetScalar(prhs[1]); // Frequency [Hz]
  float pitch = mxGetScalar(prhs[2]); // Element Pitch [m]
	float TXangle = mxGetScalar(prhs[3]); // TX Angle [rad]
	float c = mxGetScalar(prhs[4]); // Sound Speed [m/s]
	float t0 = mxGetScalar(prhs[5]); // Acquisition Start Time [s]

  // Zero-padding before FFTs
	// Time direction: extensive zero-padding is required with linear interpolation
	int ntshift = (int)(2 * ceil(t0*fs / 2));
	int ntFFT = 4 * nt + ntshift;
	// X direction: in order to avoid lateral edge effects
	float factor = 1.5f;
	int nxFFT = (int)(2 * ceil(factor*nx / 2));
	// Write values in for f0
	float* f0 = (float *)malloc(sizeof(float) * (ntFFT / 2 + 1));
	for (int i = 0; i < ntFFT / 2 + 1; i++)
		f0[i] = (float)i*fs / ntFFT;
	// Write values in for kx
	float* kx = (float *)malloc(sizeof(float) * nxFFT);
	for (int i = 0; i < nxFFT; i++)
		kx[i] = (float)((i > nxFFT / 2) ? i - nxFFT : i) / pitch / nxFFT;
	// Convert both f0 and kx to device arrays
	float *d_f0, *d_kx;
	CCE(hipMalloc(&d_f0, (ntFFT / 2 + 1) * sizeof(float)));
	CCE(hipMalloc(&d_kx, nxFFT * sizeof(float)));
	CCE(hipMemcpy(d_f0, f0, (ntFFT / 2 + 1) * sizeof(float), HtoD));
	CCE(hipMemcpy(d_kx, kx, nxFFT * sizeof(float), HtoD));

	// Read Signals Into Host Array and Copy to Device
	hipfftComplex *SIG = (hipfftComplex *)malloc(ntFFT*nxFFT*sizeof(hipfftComplex));
	for (int jj = 0; jj < ntFFT; jj++) {
		for (int ii = 0; ii < nxFFT; ii++) {
			SIG[ii + jj*nxFFT].x = 0;
			SIG[ii + jj*nxFFT].y = 0;
		}
	}
	for (int jj = 0; jj < nt; jj++) {
		for (int ii = 0; ii < nx; ii++) {
			SIG[ii + jj*nxFFT].x = (float) SIGinput[ii + jj*nx];
			SIG[ii + jj*nxFFT].y = 0;
		}
	}
	hipfftComplex *d_SIG, *d_SIG_t;
	CCE(hipMalloc(&d_SIG, ntFFT * nxFFT * sizeof(hipfftComplex)));
	CCE(hipMalloc(&d_SIG_t, ntFFT * nxFFT * sizeof(hipfftComplex)));
	CCE(hipMemcpy(d_SIG, SIG, ntFFT * nxFFT * sizeof(hipfftComplex), HtoD));

	// Take Temporal FFT
	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid((nxFFT + dimBlock.x - 1) / dimBlock.x,
		(ntFFT + dimBlock.y - 1) / dimBlock.y, 1);
	transpose << <dimGrid, dimBlock >> >(d_SIG_t, d_SIG, ntFFT, nxFFT);
	batchedFFT(d_SIG_t, ntFFT, nxFFT);
	dim3 dimBlockT(16, 16, 1);
	dim3 dimGridT((ntFFT + dimBlock.x - 1) / dimBlock.x,
		(nxFFT + dimBlock.y - 1) / dimBlock.y, 1);
	transpose << <dimGridT, dimBlockT >> >(d_SIG, d_SIG_t, nxFFT, ntFFT);

	// ERM velocity
	float sinA = sinf(TXangle);
	float cosA = cosf(TXangle);
	float v = c / sqrt(1 + cosA + sinA * sinA);

	// Compensate for steering angle and/or depth start
	// Assumes that t=0 is when any element first sends its tx
	float* dt = (float *)malloc(sizeof(float) * nx);
	for (int i = 0; i < nx; i++)
		dt[i] = (float)((TXangle < 0) ? nx - 1 - i : -i)*sinA*pitch / c;
	float *d_dt;
	CCE(hipMalloc(&d_dt, nx * sizeof(float)));
	CCE(hipMemcpy(d_dt, dt, nx * sizeof(float), hipMemcpyHostToDevice));
	rfTrim << <dimGrid, dimBlock >> >(d_SIG, ntFFT / 2 + 1, nx, nxFFT, d_dt, d_f0, t0);

	// Take Azimuthal (Spatial) FFT
	batchedFFT(d_SIG, nxFFT, ntFFT / 2 + 1);

	// Perform Stolt Mapping
	removeEvanescent << <dimGrid, dimBlock >> >(d_SIG, d_f0, ntFFT / 2 + 1, d_kx, nxFFT, c);
	// Separate real and imaginary components
	hipfftComplex *d_SIGforTexture;
	CCE(hipMalloc(&d_SIGforTexture, (ntFFT / 2 + 1) * nxFFT * sizeof(hipfftComplex)));
	transpose << <dimGrid, dimBlock >> >(d_SIGforTexture, d_SIG, ntFFT / 2 + 1, nxFFT);
	float *d_SIGreal, *d_SIGimag;
	CCE(hipMalloc(&d_SIGreal, (ntFFT / 2 + 1) * nxFFT * sizeof(float)));
	CCE(hipMalloc(&d_SIGimag, (ntFFT / 2 + 1) * nxFFT * sizeof(float)));
	dim3 dimBlockTex(16, 16, 1);
	dim3 dimGridTex((ntFFT / 2 + dimBlock.x) / dimBlock.x,
		(nxFFT + dimBlock.y - 1) / dimBlock.y, 1);
	getRealAndImag << <dimGridTex, dimBlockTex >> >(d_SIGforTexture, d_SIGreal, d_SIGimag, nxFFT, ntFFT / 2 + 1);
	// Write real and imaginary parts back to host memory
	float *SIGreal = (float *)malloc(nxFFT * (ntFFT / 2 + 1) * sizeof(float));
	float *SIGimag = (float *)malloc(nxFFT * (ntFFT / 2 + 1) * sizeof(float));
	CCE(hipMemcpy(SIGreal, d_SIGreal, nxFFT * (ntFFT / 2 + 1) * sizeof(float), DtoH));
	CCE(hipMemcpy(SIGimag, d_SIGimag, nxFFT * (ntFFT / 2 + 1) * sizeof(float), DtoH));

	// Make the Spatio-Temporal Fourier Domain of the Signals a Texture
	// Real Part
	// Create CUDA Array to Interpolate on
	hipExtent extentDescReal = make_hipExtent(ntFFT/2+1, 0, nxFFT);  // <-- 0 height required for 1D-Layered
	hipChannelFormatDesc channelDescReal = hipCreateChannelDesc<float>();
	hipMemcpy3DParms mParamsReal = { 0 };
	mParamsReal.srcPtr = make_hipPitchedPtr(SIGreal, (ntFFT / 2 + 1) * sizeof(float), ntFFT / 2 + 1, 1);
	mParamsReal.kind = hipMemcpyHostToDevice;
	mParamsReal.extent = make_hipExtent(ntFFT / 2 + 1, 1, nxFFT);
	hipArray* cuArrayReal;
	CCE(hipMalloc3DArray(&cuArrayReal, &channelDescReal, extentDescReal, hipArrayLayered));
	mParamsReal.dstArray = cuArrayReal;
	CCE(hipMemcpy3D(&mParamsReal));
	// Texture Description
	texDescReal.addressMode[0] = hipAddressModeBorder;
	texDescReal.filterMode = hipFilterModeLinear;
	texDescReal.normalizedCoords = 0; // false -- Not using normalized coords
	texDescReal.readMode = hipReadModeElementType;
	// Resource Description
	resDescReal.resType = hipResourceTypeArray;
  resDescReal.res.array.array = cuArrayReal;
	// Create CUDA Texture Object from Texture and Resource Descriptions
	CCE(hipCreateTextureObject(&texObjReal, &resDescReal, &texDescReal, NULL));
	// Imaginary Part
	// Create CUDA Array to Interpolate on
	hipExtent extentDescImag = make_hipExtent(ntFFT / 2 + 1, 0, nxFFT);  // <-- 0 height required for 1D-Layered
	hipChannelFormatDesc channelDescImag = hipCreateChannelDesc<float>();
	hipMemcpy3DParms mParamsImag = { 0 };
	mParamsImag.srcPtr = make_hipPitchedPtr(SIGimag, (ntFFT / 2 + 1) * sizeof(float), ntFFT / 2 + 1, 1);
	mParamsImag.kind = hipMemcpyHostToDevice;
	mParamsImag.extent = make_hipExtent(ntFFT / 2 + 1, 1, nxFFT);
	hipArray* cuArrayImag;
	CCE(hipMalloc3DArray(&cuArrayImag, &channelDescImag, extentDescImag, hipArrayLayered));
	mParamsImag.dstArray = cuArrayImag;
	CCE(hipMemcpy3D(&mParamsImag));
	// Texture Description
	texDescImag.addressMode[0] = hipAddressModeBorder;
	texDescImag.filterMode = hipFilterModeLinear;
	texDescReal.normalizedCoords = 0; // false -- Not using normalized coords
	texDescImag.readMode = hipReadModeElementType;
	// Resource Description
	resDescImag.resType = hipResourceTypeArray;
  resDescImag.res.array.array = cuArrayImag;
	// Create CUDA Texture Object from Texture and Resource Descriptions
	CCE(hipCreateTextureObject(&texObjImag, &resDescImag, &texDescImag, NULL));

	// Invoke Stolt Mapping Kernel
	float beta = (1 + cosA) * sqrt(1 + cosA) / (1 + cosA + sinA * sinA);
	stoltmap << <dimGridTex, dimBlockTex >> >(d_SIGforTexture, d_f0, d_kx, ntFFT, nxFFT, c, v, beta, fs, texObjReal, texObjImag);

	// Take Axial IFFT
	concatNegAxialFreq << <dimGrid, dimBlock >> >(d_SIG, d_SIGforTexture, ntFFT, nxFFT);
	transpose << <dimGrid, dimBlock >> >(d_SIG_t, d_SIG, ntFFT, nxFFT);
	batchedIFFT(d_SIG_t, ntFFT, nxFFT);
	transpose << <dimGridT, dimBlockT >> >(d_SIG, d_SIG_t, nxFFT, ntFFT);

	// Steering Angle Compensation
	float gamma = sinA / (2 - cosA);
	steerComp << <dimGrid, dimBlock >> >(d_SIG, nxFFT, ntFFT, d_kx, fs, c, gamma);

	// Take Spatial IFFT
	batchedIFFT(d_SIG, nxFFT, ntFFT);
	CCE(hipMemcpy(SIG, d_SIG, ntFFT * nxFFT * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	// Write final migrated signal to file
	plhs[0] = mxCreateDoubleMatrix( nx, nt, mxREAL);
  double *migSIG = (double *)mxGetPr(plhs[0]);
	for (int jj = 0; jj < nt; jj++) {
		for (int ii = 0; ii < nx; ii++) {
			migSIG[ii + jj*nx] = (double) SIG[ii + (jj+ntshift)*nxFFT].x;
		}
	}

	// Destroy all allocations and reset all state on the current device in the current process.
	CCE(hipDeviceReset());

}
