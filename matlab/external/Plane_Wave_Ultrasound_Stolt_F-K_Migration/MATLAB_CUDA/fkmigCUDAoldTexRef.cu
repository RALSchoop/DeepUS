#include "hip/hip_runtime.h"
// To Compile: nvcc fkmigCUDA.cu -o fkmigCUDA.out -I/usr/local/cuda/include -L/usr/local/cuda/lib -lcufft
// To Run: ./fkmigCUDA.out SIG.txt nt nx fs pitch TXangle c t0 migSIG.txt

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <typeinfo>
#include <iostream>
#include <mex.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <math.h>

// Define macros for excessively long names
#define pi 3.141592653589793238462643383279502884197169399375105820974f
#define CCE checkCudaErrors
#define HtoD hipMemcpyHostToDevice
#define DtoH hipMemcpyDeviceToHost

// 1D Layered Textures for Real and Imaginary Parts of Spatiotemporal Frequency Domain of Signals
texture<float, hipTextureType1DLayered, hipReadModeElementType> texRefReal;
texture<float, hipTextureType1DLayered, hipReadModeElementType> texRefImag;

// Runs batched FFT and IFFT on device data
void batchedFFT(hipfftComplex* dData, int N, int BATCH) {
	hipfftHandle plan;
	if (hipfftPlan1d(&plan, N, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}
	if (hipfftExecC2C(plan, dData, dData, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
	}
	if (hipDeviceSynchronize() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
	}
}
void batchedIFFT(hipfftComplex* dData, int N, int BATCH) {
	hipfftHandle plan;
	if (hipfftPlan1d(&plan, N, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}
	if (hipfftExecC2C(plan, dData, dData, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
	}
	if (hipDeviceSynchronize() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
	}
}

// Outputs Matrix Transpose
__global__ void transpose(hipfftComplex *odata, hipfftComplex *idata, int numRows, int numCols)
{
	int c_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int r_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (r_idx < numRows && c_idx < numCols) {
		float origx = idata[c_idx + numCols * r_idx].x;
		float origy = idata[c_idx + numCols * r_idx].y;
		__syncthreads();
		odata[r_idx + numRows * c_idx].x = origx;
		odata[r_idx + numRows * c_idx].y = origy;
	}
}

// Trim the RF Signals
__global__ void rfTrim(hipfftComplex *SIG, int nf, int nx, int nxFFT, float *dt, float *f0, float t0)
{
	int x_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int f_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (x_idx < nx && f_idx < nf) {
		float realSIG = SIG[x_idx + f_idx * nxFFT].x;
		float imagSIG = SIG[x_idx + f_idx * nxFFT].y;
		float phase = -2 * pi*(dt[x_idx] + t0)*f0[f_idx];
		SIG[x_idx + f_idx * nxFFT].x = realSIG * cosf(phase) - imagSIG * sinf(phase);
		SIG[x_idx + f_idx * nxFFT].y = realSIG * sinf(phase) + imagSIG * cosf(phase);
	}
}

// Remove Evanescent Parts in Spatio-temporal Frequency Domain of the Signals
__global__ void removeEvanescent(hipfftComplex *SIG, float *f0, int nf, float *kx, int nxFFT, float c)
{
	int kx_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int f0_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (kx_idx < nxFFT && f0_idx < nf) {
		if (abs(f0[f0_idx]) * abs(kx[kx_idx]) < c) {
			SIG[kx_idx + f0_idx * nxFFT].x = 0;
			SIG[kx_idx + f0_idx * nxFFT].y = 0;
		}
	}
}

// Get Real and Imaginary Parts
__global__ void getRealAndImag(hipfftComplex *cmpdata, float *realdata, float *imagdata, int numRows, int numCols)
{
	int c_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int r_idx = blockIdx.y*blockDim.y + threadIdx.y;
	if (r_idx < numRows && c_idx < numCols) {
		realdata[c_idx + numCols * r_idx] = cmpdata[c_idx + numCols * r_idx].x;
		imagdata[c_idx + numCols * r_idx] = cmpdata[c_idx + numCols * r_idx].y;
	}
}

// Run Stolt Mapping Kernel
__global__ void stoltmap(hipfftComplex *SIG, float *f0, float *kx, int ntFFT, int nxFFT, float c, float v, float beta, float fs)
{
	int f0_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int kx_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (kx_idx == 0 && f0_idx == 0) {
		SIG[0].x = 0;
		SIG[0].y = 0;
	}
	else if (kx_idx < nxFFT && f0_idx < ntFFT / 2 + 1) {
		// Note: we choose kz = 2*f/c (i.e. z = c*t/2)
		float Kx = kx[kx_idx];
		float f = f0[f0_idx];
		float fkz = v*sqrt(Kx*Kx + 4 * ((f*f) / (c*c)) / (beta*beta));
		__syncthreads();
		// Linear interpolation in the frequency domain: f -> fkz
		float fkz_idx = (fkz / (fs / ntFFT)) + 0.5f;
		float SIGreal = tex1DLayered(texRefReal, fkz_idx, kx_idx);
		float SIGimag = tex1DLayered(texRefImag, fkz_idx, kx_idx);
		__syncthreads();
		// Multiply By Obliquity factor: f / fkz
		SIG[kx_idx + f0_idx * nxFFT].x = SIGreal * f / fkz;
		SIG[kx_idx + f0_idx * nxFFT].y = SIGimag * f / fkz;
		__syncthreads();
	}
}

// Concatenate Negative Axial Frequencies to Fourier Domain of Migrated Solution
__global__ void concatNegAxialFreq(hipfftComplex *SIG, hipfftComplex *SIGfromTexture, int ntFFT, int nxFFT)
{
	int kx_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int f0_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (kx_idx < nxFFT && f0_idx < ntFFT) {
		if (f0_idx < ntFFT / 2 + 1) {
			// Original Part
			SIG[kx_idx + f0_idx * nxFFT].x = SIGfromTexture[kx_idx + f0_idx * nxFFT].x;
			SIG[kx_idx + f0_idx * nxFFT].y = SIGfromTexture[kx_idx + f0_idx * nxFFT].y;
		}
		else {
			// Concatenated Part
			SIG[kx_idx + f0_idx * nxFFT].x = SIGfromTexture[((nxFFT - kx_idx) % nxFFT) + (ntFFT - f0_idx) * nxFFT].x;
			SIG[kx_idx + f0_idx * nxFFT].y = -SIGfromTexture[((nxFFT - kx_idx) % nxFFT) + (ntFFT - f0_idx) * nxFFT].y;
		}
	}
}

// Steering Angle Compensation for RF Signals
__global__ void steerComp(hipfftComplex *SIG, int nxFFT, int ntFFT, float *kx, float fs, float c, float gamma)
{
	int kx_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int t_idx = blockIdx.y*blockDim.y + threadIdx.y;

	if (kx_idx < nxFFT && t_idx < ntFFT) {
		float realSIG = SIG[kx_idx + t_idx * nxFFT].x;
		float imagSIG = SIG[kx_idx + t_idx * nxFFT].y;
		float dx = -gamma*t_idx / fs*c / 2;
		float phase = -2 * pi * kx[kx_idx] * dx;
		SIG[kx_idx + t_idx * nxFFT].x = realSIG * cosf(phase) - imagSIG * sinf(phase);
		SIG[kx_idx + t_idx * nxFFT].y = realSIG * sinf(phase) + imagSIG * cosf(phase);
	}
}


// Gateway function
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

	hipDeviceReset(); // Reset Everything on the GPU

	// Argument check
	if (nrhs != 6)	{ mexErrMsgTxt("Wrong number of inputs.\n"); }
  if (nlhs != 1)	{ mexErrMsgTxt("Wrong number of outputs.\n"); }

	// Create CUDA Err
  hipError_t e;

  // Gather values from inputs
  double *SIGinput = (double *)mxGetData(prhs[0]);
  const mwSize *dimsSIG = mxGetDimensions(prhs[0]);
  mwSize nx = dimsSIG[0]; // Number of Array Elements [ROWS]
  mwSize nt = dimsSIG[1]; // Number of Time Points [COLUMNS]
  float fs = mxGetScalar(prhs[1]); // Frequency [Hz]
  float pitch = mxGetScalar(prhs[2]); // Element Pitch [m]
	float TXangle = mxGetScalar(prhs[3]); // TX Angle [rad]
	float c = mxGetScalar(prhs[4]); // Sound Speed [m/s]
	float t0 = mxGetScalar(prhs[5]); // Acquisition Start Time [s]
	mexPrintf("Loaded all Inputs:\nfs: %f\npitch: %f\nTXangle: %f\nc: %f\nt0: %f\n\n", fs, pitch, TXangle, c, t0);
	mexPrintf("Input Signals:\nNumber of Elements: %d\nNumber of Time Points: %d\n\n", nx, nt);

  // Zero-padding before FFTs
	// Time direction: extensive zero-padding is required with linear interpolation
	int ntshift = (int)(2 * ceil(t0*fs / 2));
	int ntFFT = 4 * nt + ntshift;
	// X direction: in order to avoid lateral edge effects
	float factor = 1.5f;
	int nxFFT = (int)(2 * ceil(factor*nx / 2));
	mexPrintf("ntFFT: %d\nnxFFT: %d\n\n", ntFFT, nxFFT);
	// Write values in for f0
	float* f0 = (float *)malloc(sizeof(float) * (ntFFT / 2 + 1));
	for (int i = 0; i < ntFFT / 2 + 1; i++)
		f0[i] = (float)i*fs / ntFFT;
	// Write values in for kx
	float* kx = (float *)malloc(sizeof(float) * nxFFT);
	for (int i = 0; i < nxFFT; i++)
		kx[i] = (float)((i > nxFFT / 2) ? i - nxFFT : i) / pitch / nxFFT;
	// Convert both f0 and kx to device arrays
	float *d_f0, *d_kx;
	CCE(hipMalloc(&d_f0, (ntFFT / 2 + 1) * sizeof(float)));
	CCE(hipMalloc(&d_kx, nxFFT * sizeof(float)));
	CCE(hipMemcpy(d_f0, f0, (ntFFT / 2 + 1) * sizeof(float), HtoD));
	CCE(hipMemcpy(d_kx, kx, nxFFT * sizeof(float), HtoD));
	mexPrintf("Zero-padding before FFTs\n");

	// Read Signals Into Host Array and Copy to Device
	hipfftComplex *SIG = (hipfftComplex *)malloc(ntFFT*nxFFT*sizeof(hipfftComplex));
	for (int jj = 0; jj < ntFFT; jj++) {
		for (int ii = 0; ii < nxFFT; ii++) {
			SIG[ii + jj*nxFFT].x = 0;
			SIG[ii + jj*nxFFT].y = 0;
		}
	}
	for (int jj = 0; jj < nt; jj++) {
		for (int ii = 0; ii < nx; ii++) {
			SIG[ii + jj*nxFFT].x = (float) SIGinput[ii + jj*nx];
			SIG[ii + jj*nxFFT].y = 0;
		}
	}
	hipfftComplex *d_SIG, *d_SIG_t;
	CCE(hipMalloc(&d_SIG, ntFFT * nxFFT * sizeof(hipfftComplex)));
	CCE(hipMalloc(&d_SIG_t, ntFFT * nxFFT * sizeof(hipfftComplex)));
	CCE(hipMemcpy(d_SIG, SIG, ntFFT * nxFFT * sizeof(hipfftComplex), HtoD));

	e = hipGetLastError();
	if (e) mexPrintf("Error After Reading Signals and Writing to Device Array: %d %s\n", e, hipGetErrorString(e));

	mexPrintf("Signals Read Into Host Array and Copied to Device\n");

	// Take Temporal FFT
	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid((nxFFT + dimBlock.x - 1) / dimBlock.x,
		(ntFFT + dimBlock.y - 1) / dimBlock.y, 1);
	transpose << <dimGrid, dimBlock >> >(d_SIG_t, d_SIG, ntFFT, nxFFT);
	batchedFFT(d_SIG_t, ntFFT, nxFFT);
	dim3 dimBlockT(16, 16, 1);
	dim3 dimGridT((ntFFT + dimBlock.x - 1) / dimBlock.x,
		(nxFFT + dimBlock.y - 1) / dimBlock.y, 1);
	transpose << <dimGridT, dimBlockT >> >(d_SIG, d_SIG_t, nxFFT, ntFFT);

	e = hipGetLastError();
	if (e) mexPrintf("Error After Temporal FFT: %d %s\n", e, hipGetErrorString(e));

	mexPrintf("Temporal FFT Complete\n");

	// ERM velocity
	float sinA = sinf(TXangle);
	float cosA = cosf(TXangle);
	float v = c / sqrt(1 + cosA + sinA * sinA);

	// Compensate for steering angle and/or depth start
	// Assumes that t=0 is when any element first sends its tx
	float* dt = (float *)malloc(sizeof(float) * nx);
	for (int i = 0; i < nx; i++)
		dt[i] = (float)((TXangle < 0) ? nx - 1 - i : -i)*sinA*pitch / c;
	float *d_dt;
	CCE(hipMalloc(&d_dt, nx * sizeof(float)));
	CCE(hipMemcpy(d_dt, dt, nx * sizeof(float), hipMemcpyHostToDevice));
	rfTrim << <dimGrid, dimBlock >> >(d_SIG, ntFFT / 2 + 1, nx, nxFFT, d_dt, d_f0, t0);

	e = hipGetLastError();
	if (e) printf("Error After RF Trimming: %d %s\n", e, hipGetErrorString(e));


	// Take Azimuthal (Spatial) FFT
	batchedFFT(d_SIG, nxFFT, ntFFT / 2 + 1);

	e = hipGetLastError();
	if (e) printf("Error After Azimuthal FFT: %d %s\n", e, hipGetErrorString(e));

	mexPrintf("Steering Angle Compensation Complete\n");


	// Perform Stolt Mapping
	removeEvanescent << <dimGrid, dimBlock >> >(d_SIG, d_f0, ntFFT / 2 + 1, d_kx, nxFFT, c);
	// Separate real and imaginary components
	hipfftComplex *d_SIGforTexture;
	CCE(hipMalloc(&d_SIGforTexture, (ntFFT / 2 + 1) * nxFFT * sizeof(hipfftComplex)));
	transpose << <dimGrid, dimBlock >> >(d_SIGforTexture, d_SIG, ntFFT / 2 + 1, nxFFT);
	float *d_SIGreal, *d_SIGimag;
	CCE(hipMalloc(&d_SIGreal, (ntFFT / 2 + 1) * nxFFT * sizeof(float)));
	CCE(hipMalloc(&d_SIGimag, (ntFFT / 2 + 1) * nxFFT * sizeof(float)));
	dim3 dimBlockTex(16, 16, 1);
	dim3 dimGridTex((ntFFT / 2 + dimBlock.x) / dimBlock.x,
		(nxFFT + dimBlock.y - 1) / dimBlock.y, 1);
	getRealAndImag << <dimGridTex, dimBlockTex >> >(d_SIGforTexture, d_SIGreal, d_SIGimag, nxFFT, ntFFT / 2 + 1);
	// Write real and imaginary parts back to host memory
	float *SIGreal = (float *)malloc(nxFFT * (ntFFT / 2 + 1) * sizeof(float));
	float *SIGimag = (float *)malloc(nxFFT * (ntFFT / 2 + 1) * sizeof(float));
	CCE(hipMemcpy(SIGreal, d_SIGreal, nxFFT * (ntFFT / 2 + 1) * sizeof(float), DtoH));
	CCE(hipMemcpy(SIGimag, d_SIGimag, nxFFT * (ntFFT / 2 + 1) * sizeof(float), DtoH));

	// Make the Spatio-Temporal Fourier Domain of the Signals a Texture
	// Real Part
	hipExtent extentDescReal = make_hipExtent(ntFFT/2+1, 0, nxFFT);  // <-- 0 height required for 1Dlayered
	hipChannelFormatDesc channelDescReal = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMemcpy3DParms mParamsReal = { 0 };
	mParamsReal.srcPtr = make_hipPitchedPtr(SIGreal, (ntFFT / 2 + 1) * sizeof(float), ntFFT / 2 + 1, 1);
	mParamsReal.kind = hipMemcpyHostToDevice;
	mParamsReal.extent = make_hipExtent(ntFFT / 2 + 1, 1, nxFFT);
	hipArray* cuArrayReal;
	hipMalloc3DArray(&cuArrayReal, &channelDescReal, extentDescReal, hipArrayLayered);
	mParamsReal.dstArray = cuArrayReal;
	hipMemcpy3D(&mParamsReal);
	texRefReal.addressMode[0] = hipAddressModeBorder;
	texRefReal.filterMode = hipFilterModeLinear;
	texRefReal.normalized = false;
	hipBindTextureToArray(texRefReal, cuArrayReal, channelDescReal);

	// Imaginary Part
	hipExtent extentDescImag = make_hipExtent(ntFFT / 2 + 1, 0, nxFFT);  // <-- 0 height required for 1Dlayered
	hipChannelFormatDesc channelDescImag = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMemcpy3DParms mParamsImag = { 0 };
	mParamsImag.srcPtr = make_hipPitchedPtr(SIGimag, (ntFFT / 2 + 1) * sizeof(float), ntFFT / 2 + 1, 1);
	mParamsImag.kind = hipMemcpyHostToDevice;
	mParamsImag.extent = make_hipExtent(ntFFT / 2 + 1, 1, nxFFT);
	hipArray* cuArrayImag;
	hipMalloc3DArray(&cuArrayImag, &channelDescImag, extentDescImag, hipArrayLayered);
	mParamsImag.dstArray = cuArrayImag;
	hipMemcpy3D(&mParamsImag);
	texRefImag.addressMode[0] = hipAddressModeBorder;
	texRefImag.filterMode = hipFilterModeLinear;
	texRefImag.normalized = false;
	hipBindTextureToArray(texRefImag, cuArrayImag, channelDescImag);


	// Invoke Stolt Mapping Kernel
	float beta = (1 + cosA) * sqrt(1 + cosA) / (1 + cosA + sinA * sinA);
	stoltmap << <dimGridTex, dimBlockTex >> >(d_SIGforTexture, d_f0, d_kx, ntFFT, nxFFT, c, v, beta, fs);

	e = hipGetLastError();
	if (e) mexPrintf("Error After Stolt Mapping: %d %s\n", e, hipGetErrorString(e));

	mexPrintf("Stolt Mapping Complete\n");


	// Take Axial IFFT
	concatNegAxialFreq << <dimGrid, dimBlock >> >(d_SIG, d_SIGforTexture, ntFFT, nxFFT);
	transpose << <dimGrid, dimBlock >> >(d_SIG_t, d_SIG, ntFFT, nxFFT);
	batchedIFFT(d_SIG_t, ntFFT, nxFFT);
	transpose << <dimGridT, dimBlockT >> >(d_SIG, d_SIG_t, nxFFT, ntFFT);

	e = hipGetLastError();
	if (e) mexPrintf("Error After Axial IFFT: %d %s\n", e, hipGetErrorString(e));

	mexPrintf("Axial IFFT Complete\n");


	// Steering Angle Compensation
	float gamma = sinA / (2 - cosA);
	steerComp << <dimGrid, dimBlock >> >(d_SIG, nxFFT, ntFFT, d_kx, fs, c, gamma);

	e = hipGetLastError();
	if (e) mexPrintf("Error After Steering Angle Compensation: %d %s\n", e, hipGetErrorString(e));

	mexPrintf("Steering Angle Compensation\n");


	// Take Spatial IFFT
	batchedIFFT(d_SIG, nxFFT, ntFFT);
	CCE(hipMemcpy(SIG, d_SIG, ntFFT * nxFFT * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	e = hipGetLastError();
	if (e) mexPrintf("Error After Spatial IFFT: %d %s\n", e, hipGetErrorString(e));

	mexPrintf("Spatial IFFT Complete\n");


	// Write final migrated signal to file
	plhs[0] = mxCreateDoubleMatrix( nx, nt, mxREAL);
  double *migSIG = (double *)mxGetPr(plhs[0]);
	for (int jj = 0; jj < nt; jj++) {
		for (int ii = 0; ii < nx; ii++) {
			migSIG[ii + jj*nx] = (double) SIG[ii + (jj+ntshift)*nxFFT].x;
		}
	}


	// Free all allocated memory
	hipFree(d_SIG);
	hipFree(d_SIG_t);
	hipFree(d_SIGforTexture);
	hipFree(d_SIGreal);
	hipFree(d_SIGimag);


}
